//
// Created by andwh on 04/11/2024.
//

using namespace std;

#include "main.cuh"

#include <chrono>
#include "engine/Domain.h"
#include "automata_parser/uppaal_xml_parser.h"
#include <iostream>
#include "output.h"
#include "network_optimization/domain_optimization_visitor.h"
#include "network_optimization/pn_compile_visitor.h"
#include "simulation/simulation.cuh"
#include "simulation/simulation_config.h"
#include "simulation/state/SharedModelState.cuh"
#include "automata_parser/VariableUsageVisitor.h"

#include "simulation/simulation.cuh"

VariableKind* createKindArray(const std::unordered_map<int, VariableTrackingVisitor::VariableUsage>& registry) {
    VariableKind* kinds = new VariableKind[registry.size()];
    for(int i = 0; i < registry.size(); i++) {
        kinds[i] = registry.at(i).kind;
    }
    return kinds;
}


int main()
{
    // Hardcoded path to the XML file
    std::string filename = "../automata_parser/XmlFiles/agentBaseCovid_100_1.0.xml";
    string query1 = "c2.sdh";
    string query2 = "c2.g4sdgh";
    std::unordered_set<std::string>* query_set = new std::unordered_set<std::string>();
    query_set->insert(query1);
    query_set->insert(query2);

    abstract_parser* parser = new uppaal_xml_parser();

    if constexpr (VERBOSE) {std::cout << "Test after instantiate_parser" << std::endl;}

    network model = parser->parse(filename);

    if constexpr (VERBOSE) {std::cout << "Parsing successful. Network details:" << std::endl;}

    if constexpr (VERBOSE) {cout << "Performing optimizations..." << endl;}

    network_props properties = {};
    simulation_config config = {};

    auto sim = simulation(parser);

    properties.node_edge_map = new std::unordered_map<int, std::list<edge>>(parser->get_node_edge_map());
    properties.start_nodes = new std::list<int>(parser->get_start_nodes());
    properties.template_names = new std::unordered_map<int, std::string>(*parser->get_template_names());
    properties.variable_names = new std::unordered_map<int, std::string>(*parser->get_clock_names());    // this can create mem leaks.
    properties.node_network = new std::unordered_map<int, int>(*parser->get_subsystems());
    properties.node_names = new std::unordered_map<int, std::string>(*parser->get_nodes_with_name());



    std::unordered_map<std::string, int> template_name_int_map;
    for (auto itr = properties.template_names->cbegin(); itr != properties.template_names->cend(); itr++) {
        template_name_int_map.insert({itr->second, itr->first});
    }

    std::unordered_map<std::string, int> node_name_int_map;
    for (auto itr = properties.node_names->cbegin(); itr != properties.node_names->cend(); itr++) {
        node_name_int_map.insert({itr->second, itr->first});
    }



    domain_optimization_visitor optimizer = domain_optimization_visitor(
        query_set,
        properties.node_network,
        properties.node_names,
        properties.template_names);
    optimizer.optimize(&model);

    pn_compile_visitor pn_compiler;
    pn_compiler.visit(&model);

    setup_simulation_config(&config, &model, optimizer.get_max_expr_depth(), optimizer.get_max_fanout(), optimizer.get_node_count());

    VariableTrackingVisitor var_tracker;
    var_tracker.visit(&model);
    if constexpr (VERBOSE) {
        var_tracker.print_variable_usage();
    }
    auto registry = var_tracker.get_variable_registry();

    VariableKind* kinds = createKindArray(registry);
    int num_vars = registry.size();

    if constexpr (VERBOSE) {
        for(int i = 0; i < registry.size(); i++) {
            printf("Kind %d: %d\n", i, kinds[i]);
        }
    }

    // Simulation count
    size_t simulations = config.total_simulations();

    // Query analysis loop (Sidenote: Fuck unordered sets)
    for (auto itr = query_set->cbegin(); itr != query_set->cend(); itr++){
        // Query string
        string query = *(*query_set).find(*itr);

        if constexpr (VERBOSE){
            cout << "Recorded query is: " + query << endl;
        }


        // String split
        std::vector<char> component;
        std::vector<char> goal_node;
        bool period_reached = false;
        for (int i = 0; i < query.length(); i++){
            // Guard
            if (query[i] == '.'){period_reached = true; continue;}
            if (!period_reached) {
                component.push_back(query[i]);
            }
            if (period_reached) {
                goal_node.push_back(query[i]);
            }
        }
        std::string component_name(component.begin(), component.end());
        //template_name_int_map.find(component_name);

        std::string node_name(goal_node.begin(), goal_node.end());
        auto temp = node_name_int_map.find(node_name);
        std::unordered_map<int, node*> node_map = optimizer.get_node_map();

        if (temp != node_name_int_map.cend()) {
            int goal_node_idx = (*temp).second;

            (*node_map.find(goal_node_idx)).second->type = node::goal;
        }

        SharedModelState* state = init_shared_model_state(
            &model,  // cpu_network
            *optimizer.get_node_subsystems_map(),
            *properties.node_edge_map,
            node_map,
            var_tracker.get_variable_registry(),
            parser,
            num_vars);


        bool* goal_flags_host_ptr = (bool*)malloc(simulations*sizeof(bool));
        for (int i = 0; i < simulations*sizeof(bool); i++) {
            goal_flags_host_ptr[i] = false;
        }

        bool* goal_flags_device_ptr;
        hipMalloc(&goal_flags_device_ptr, simulations*sizeof(bool));

        hipMemcpy(goal_flags_device_ptr, goal_flags_host_ptr, simulations*sizeof(bool), hipMemcpyHostToDevice);

        // Run the SMC simulations
        sim.run_statistical_model_checking(state, 0.05, 0.01, kinds, num_vars, goal_flags_device_ptr);

        hipMemcpy(goal_flags_host_ptr, goal_flags_device_ptr, simulations*sizeof(bool), hipMemcpyDeviceToHost);
        cout << hipGetErrorString(hipGetLastError()) << endl;
        int hits = 0;
        for (int i = 0; i < simulations*sizeof(bool); i++) {
            if (goal_flags_host_ptr[i]) {hits += 1;}
        }

        free(goal_flags_host_ptr);
        hipFree(goal_flags_device_ptr);

        cout << "Total number of simulations: " + std::to_string(simulations) << endl;

        float res = hits / simulations;

        string output = "The answer to " + query + " is " + std::to_string(res);

        cout << output << endl;
    }

    // Kernels for debugging purposes
    if constexpr (VERBOSE) {
        // verify_expressions_kernel<<<1,1>>>(state);
        // hipDeviceSynchronize();
        // verify_invariants_kernel<<<1, 1>>>(state);
        // test_kernel<<<1, 1>>>(state);
        // validate_edge_indices<<<1, 1>>>(state);
    }
    // hipMalloc();
    delete parser;

    return 0;
}
