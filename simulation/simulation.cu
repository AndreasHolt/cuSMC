#include "hip/hip_runtime.h"
// Created by andwh on 24/10/2024.

#include "simulation.cuh"
#include <cmath>

#include "state/SharedModelState.cuh"
#include "state/SharedRunState.cuh"

#define NUM_RUNS 6
// #define NUM_COMPONENTS 2
// #define MAX_COMPONENTS 100
#define TIME_BOUND 1.0

#define MAX_VARIABLES 5


// Et array af locations for et specifikt component
// En funktion der mapper en værdi i det array til den relevante node

__device__ void check_cuda_error(const char* location) {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error at %s: %s\n", location, hipGetErrorString(error));
    }
}

#define CHECK_ERROR(loc) check_cuda_error(loc)


__device__ void compute_possible_delay(
    ComponentState* my_state,
    SharedBlockMemory* shared,
    SharedModelState* model,
    BlockSimulationState* block_state)  // Add this
{
    // Add debug prints
    printf("Thread %d: Computing delay for node %d\n",
           threadIdx.x, my_state->current_node->id);

    my_state->has_delay = false;

    const NodeInfo& node = *my_state->current_node;

    // Debug print invariants
    printf("Thread %d: Node has %d invariants starting at index %d\n",
           threadIdx.x, node.num_invariants, node.first_invariant_index);

    // Rest same as before but with more debug prints
    if (node.type == node::urgent || node.type == node::committed) {
        printf("Thread %d: Urgent/committed node - delay=0\n", threadIdx.x);
        my_state->next_delay = 0.0;
        my_state->has_delay = true;
        return;
    }

    double min_delay = 0.0;
    double max_delay = DBL_MAX;
    bool is_bounded = false;

    for(int i = 0; i < node.num_invariants; i++) {
        const GuardInfo& inv = model->invariants[node.first_invariant_index + i];
        if(inv.uses_variable) {
            const auto& var = shared->variables[inv.var_info.variable_id];
            printf("Thread %d: Checking invariant %d for variable %d (value=%f, rate=%d)\n",
                   threadIdx.x, i, inv.var_info.variable_id, var.value, var.rate);

            double curr_val = var.value;
            double rate = var.rate;

            if(rate == 0) {
                printf("Thread %d: Skipping invariant %d - rate is 0\n",
                       threadIdx.x, i);
                continue;
            }

            if(inv.operand == constraint::less_equal_c ||
               inv.operand == constraint::less_c) {
                double bound = 5.0; // TODO: evaluate expression
                double time_to_bound = (bound - curr_val) / rate;
                if(inv.operand == constraint::less_c) {
                    time_to_bound -= 1e-6;
                }
                printf("Thread %d: Invariant %d gives bound %f\n",
                       threadIdx.x, i, time_to_bound);

                max_delay = min(max_delay, time_to_bound);
                is_bounded = true;
            }
        }
    }

    if(is_bounded && min_delay < max_delay) {
        hiprandState* rng = block_state->random;
        double rand = hiprand_uniform(rng);
        my_state->next_delay = min_delay + (max_delay - min_delay) * rand;
        my_state->has_delay = true;
        printf("Thread %d: Sampled delay %f (min=%f, max=%f, rand=%f)\n",
               threadIdx.x, my_state->next_delay, min_delay, max_delay, rand);
    } else {
        printf("Thread %d: No delay computed (bounded=%d, min=%f, max=%f)\n",
               threadIdx.x, is_bounded, min_delay, max_delay);
    }
}




__device__ double find_minimum_delay(
    ComponentState* my_state,
    SharedBlockMemory* shared,
    const int num_components)
{
    // Each thread stores its delay in shared memory array
    __shared__ double delays[MAX_COMPONENTS];
    __shared__ int component_indices[MAX_COMPONENTS];

    // Store my delay if I have one
    if(my_state->has_delay) {
        delays[threadIdx.x] = my_state->next_delay;
        component_indices[threadIdx.x] = my_state->component_id;
    } else {
        delays[threadIdx.x] = DBL_MAX;
        component_indices[threadIdx.x] = -1;
    }
    __syncthreads();

    // Parallel reduction to find minimum
    for(int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if(threadIdx.x < stride) {
            if(delays[threadIdx.x + stride] < delays[threadIdx.x]) {
                delays[threadIdx.x] = delays[threadIdx.x + stride];
                component_indices[threadIdx.x] = component_indices[threadIdx.x + stride];
            }
        }
        __syncthreads();
    }

    // Result is in delays[0] and winning component in component_indices[0]
    return delays[0];
}


int get_total_runs(float confidence, float precision) {
    // confidence level = alpha, i.e. 0.05 for 95% confidence
    // precision = epsilon, i.e. 0.01 for +-1% error

    // int total_runs = (int)ceil(log(2.0/confidence)/log(2.0*precision*precision));
    // int total_runs = static_cast<int>(ceil(log(2.0 / confidence) / log(2.0 * precision * precision)));
    int total_runs = static_cast<size_t>(ceil((log(2.0) - log(confidence)) / (2*pow(precision, 2))));
    return total_runs;
}

// TODO: what if we want to spawn 50 trains? How do we do that?


__global__ void simulation_kernel(SharedModelState* model, bool* results,
                                int runs_per_block, float time_bound) {
    printf("Starting kernel: block=%d, thread=%d\n",
           blockIdx.x, threadIdx.x);
    CHECK_ERROR("kernel start");

    // Verify model pointer
    if(model == nullptr) {
        printf("Thread %d: NULL model pointer!\n", threadIdx.x);
        return;
    }

    __shared__ SharedBlockMemory shared_mem;
    __shared__ ComponentState components[MAX_COMPONENTS];
    __shared__ hiprandState rng_states[MAX_COMPONENTS];

    CHECK_ERROR("after shared memory declaration");

    // Debug model access
    printf("Thread %d: Attempting to access model, num_components=%d\n",
           threadIdx.x, model->num_components);
    CHECK_ERROR("after model access");

    // Setup block state
    BlockSimulationState block_state;
    block_state.model = model;
    block_state.shared = &shared_mem;
    block_state.my_component = &components[threadIdx.x];

    printf("Thread %d: Block state setup complete\n", threadIdx.x);
    CHECK_ERROR("after block state setup");

    // Initialize RNG
    int sim_id = blockIdx.x * runs_per_block;
    int comp_id = threadIdx.x;
    hiprand_init(1234 + sim_id * blockDim.x + comp_id, 0, 0,
                &rng_states[threadIdx.x]);
    block_state.random = &rng_states[threadIdx.x];

    printf("Thread %d: RNG initialized\n", threadIdx.x);
    CHECK_ERROR("after RNG init");

    // Initialize shared state
    if (threadIdx.x == 0) {
        printf("Block %d: Initializing shared memory\n", blockIdx.x);
        SharedBlockMemory::init(&shared_mem, sim_id);
    }
    __syncthreads();
    CHECK_ERROR("after shared memory init");




    // Initialize component state
    if(threadIdx.x >= model->num_components) {
        printf("Thread %d: Exiting - thread ID exceeds number of components\n",
               threadIdx.x);
        return;
    }

    ComponentState* my_state = block_state.my_component;
    my_state->component_id = comp_id;
    my_state->current_node = &model->nodes[comp_id];
    my_state->has_delay = false;

    printf("Thread %d: Component initialized, node_id=%d\n",
           threadIdx.x, my_state->current_node->id);
    CHECK_ERROR("after component init");

    // Main simulation loop
    while(shared_mem.global_time < time_bound) {
        printf("Thread %d: Time=%f\n", threadIdx.x, shared_mem.global_time);

        compute_possible_delay(my_state, &shared_mem, model, &block_state);
        CHECK_ERROR("after compute delay");
        __syncthreads();

        double min_delay = find_minimum_delay(my_state, &shared_mem, blockDim.x);
        CHECK_ERROR("after find minimum");
        printf("Thread %d: Minimum delay = %f\n", threadIdx.x, min_delay);

        if(threadIdx.x == 0) {
            shared_mem.global_time += min_delay;
            printf("Block %d: Advanced time to %f\n",
                   blockIdx.x, shared_mem.global_time);
        }
        __syncthreads();
    }

    printf("Thread %d: Simulation complete\n", threadIdx.x);
}




void simulation::run_statistical_model_checking(SharedModelState* model, float confidence, float precision) {
    int total_runs = 1;
    cout << "total_runs = " << total_runs << endl;

    // Detailed model validation
    if(model == nullptr) {
        cout << "Error: NULL model pointer" << endl;
        return;
    }

    // Print model pointer address
    cout << "Model pointer address: " << model << endl;

    // Try to access model components safely
    hipError_t error;
    SharedModelState host_model;
    error = hipMemcpy(&host_model, model, sizeof(SharedModelState), hipMemcpyDeviceToHost);
    if(error != hipSuccess) {
        cout << "CUDA error copying model: " << hipGetErrorString(error) << endl;
        return;
    }

    cout << "Model components: " << host_model.num_components << endl;

    // Print more model details
    cout << "Component sizes array at: " << host_model.component_sizes << endl;
    cout << "Nodes array at: " << host_model.nodes << endl;
    cout << "Edges array at: " << host_model.edges << endl;

    bool* device_results;
    error = hipMalloc(&device_results, total_runs * sizeof(bool));
    if(error != hipSuccess) {
        cout << "CUDA malloc error: " << hipGetErrorString(error) << endl;
        return;
    }

    // Launch configuration
    int threads_per_block = 8;
    int runs_per_block = 1;
    int num_blocks = 1;

    cout << "Launching kernel with configuration:" << endl;
    cout << "  Blocks: " << num_blocks << endl;
    cout << "  Threads per block: " << threads_per_block << endl;
    cout << "  Time bound: " << TIME_BOUND << endl;

    // Launch kernel
    simulation_kernel<<<num_blocks, threads_per_block>>>(
        model, device_results, runs_per_block, TIME_BOUND);

    error = hipGetLastError();
    if(error != hipSuccess) {
        cout << "Kernel launch error: " << hipGetErrorString(error) << endl;
        return;
    }

    cout << "Kernel launched successfully, waiting for completion..." << endl;

    error = hipDeviceSynchronize();
    if(error != hipSuccess) {
        cout << "Kernel execution error: " << hipGetErrorString(error) << endl;
        return;
    }

    cout << "Kernel execution complete" << endl;
}






__global__ void findSmallestElementInArray(float *input, int input_length, float *result, int nblocks) {
    int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    int nThreads = blockDim.x * nblocks;
    for (int i = 0; i < ceil(log2f(nThreads)); i++) { //1
        if (threadid % static_cast<int>(pow(2, i+1)==0)) {
            int correspondant = min(static_cast<int>(threadid + pow(2, i)), input_length-1);
            input[threadid] = min(input[threadid], input[correspondant]);
        }
    }
    if (threadid == 0) {
        *result = input[0];
    }
}

// void testFunction () {
//     float* h_a = new float[NUM_RUNS];
//     srand( static_cast<unsigned>(time(NULL)));
//     int upper = 3500;
//     int lower = 1230;
//     for (int i = 0; i < NUM_RUNS; i++) {
//
//         h_a[i] = rand() % (upper - lower) + lower;
//         cout << h_a[i] << ", ";
//     }
//     cout << endl;
//
//     float* d_a;
//     float d_result;
//     hipMalloc(&d_a, NUM_RUNS * sizeof(float));
//
//     hipMemcpy(d_a, h_a, NUM_RUNS * sizeof(float), hipMemcpyHostToDevice);
//
//     findSmallestElementInArray<<<1, 128>>>(d_a, NUM_RUNS, &d_result, 1); // 2 blocks (component size), 100 simulations but round up to 128
//
//     hipMemcpy(h_a, d_a, NUM_RUNS * sizeof(float), hipMemcpyDeviceToHost);
//
//     for (int i = 0; i < NUM_RUNS; i++) {
//         cout << h_a[i] << ", ";
//     }
//     cout <<endl << "Result = " << d_result << endl;
// }

void simulation::runSimulation() {
    // Problem with models, spawning new components Trains in train gate for example?
    // componentSimulation<<<NUM_COMPONENTS, 128>>>(); // 2 blocks (component size), 100 simulations but round up to 128

    // testFunction();

    // Pick delays: implement delay function
    // Find the smallest delay, and which index it has (to find component it belongs to)
    // Apply the delay
    // Pick a transition from the component that won: Pick according to the weights
    // Check whether we need to synchronize with anything when taking this transition
    // Take the transition

    // We need the state such that we can describe the run afterwards. We add our delays to it.



    cout << "test from run sim" << endl;
}
