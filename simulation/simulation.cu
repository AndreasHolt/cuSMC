#include "hip/hip_runtime.h"
// Created by andwh on 24/10/2024.

#include "simulation.cuh"
#include <cmath>

#define NUM_RUNS 6
#define NUM_COMPONENTS 2
#define MAX_COMPONENTS 100

// Et array af locations for et specifikt component
// En funktion der mapper en værdi i det array til den relevante node


// struct ComponentState {
//     struct RunState {
//         float current_time;
//         bool needs_sync;
//         float proposed_delay; // The sampled delay
//     };
//
//     __shared__ struct {
//         unordered_map<int, std::list<edge> > component_node_edge_map;
//         int component_start_node;
//         string template_name;
//     } component_data;
// };
//
// struct SimulationState {
//     ComponentState component_states[NUM_COMPONENTS][NUM_RUNS];
//
//     RunState run_states[NUM_RUNS];
// };


// __global__ void componentSimulation(SimulationState state) {
//     int component_id = blockIdx.x;
//     int run_id = threadIdx.x;
//
//     if (run_id >= NUM_COMPONENTS) return;
//
//     __shared__ ComponentInfo component_info;
//
//     if (threadIdx.x == 0) {
//         loadComponentInfo(&component_info, component_id);
//     }
//     __syncthreads(); // sync all the threads
//
//     while (!isSimulationDone(run_id)) {
//         // sim logic
//     }
//

//
//
//
//
// }




__global__ void findSmallestElementInArray(float *input, int input_length, float *result, int nblocks) {
    int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    int nThreads = blockDim.x * nblocks;
    for (int i = 0; i < ceil(log2f(nThreads)); i++) { //1
        if (threadid % static_cast<int>(pow(2, i+1)==0)) {
            int correspondant = min(static_cast<int>(threadid + pow(2, i)), input_length-1);
            input[threadid] = min(input[threadid], input[correspondant]);
        }
    }
    if (threadid == 0) {
        *result = input[0];
    }

}
void testFunction () {
    float* h_a = new float[NUM_RUNS];
    srand( static_cast<unsigned>(time(NULL)));
    int upper = 3500;
    int lower = 1230;
    for (int i = 0; i < NUM_RUNS; i++) {

        h_a[i] = rand() % (upper - lower) + lower;
        cout << h_a[i] << ", ";
    }
    cout << endl;

    float* d_a;
    float d_result;
    hipMalloc(&d_a, NUM_RUNS * sizeof(float));

    hipMemcpy(d_a, h_a, NUM_RUNS * sizeof(float), hipMemcpyHostToDevice);

    findSmallestElementInArray<<<1, 128>>>(d_a, NUM_RUNS, &d_result, 1); // 2 blocks (component size), 100 simulations but round up to 128

    hipMemcpy(h_a, d_a, NUM_RUNS * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < NUM_RUNS; i++) {
        cout << h_a[i] << ", ";
    }
    cout <<endl << "Result = " << d_result << endl;
}

void simulation::runSimulation() {
    // Problem with models, spawning new components Trains in train gate for example?
    // componentSimulation<<<NUM_COMPONENTS, 128>>>(); // 2 blocks (component size), 100 simulations but round up to 128

    // testFunction();

    // Pick delays: implement delay function
    // Find the smallest delay, and which index it has (to find component it belongs to)
    // Apply the delay
    // Pick a transition from the component that won: Pick according to the weights
    // Check whether we need to synchronize with anything when taking this transition
    // Take the transition

    // We need the state such that we can describe the run afterwards. We add our delays to it.



    cout << "test from run sim" << endl;
}
