#include "hip/hip_runtime.h"
// Created by andwh on 24/10/2024.

#include "simulation.cuh"
#include <cmath>

#include "state/SharedModelState.cuh"
#include "state/SharedRunState.cuh"

#define NUM_RUNS 6
#define TIME_BOUND 100.0

#define MAX_VARIABLES 8


// Et array af locations for et specifikt component
// En funktion der mapper en værdi i det array til den relevante node

__device__ double evaluate_expression(const expr* e, BlockSimulationState* block_state) {
    if(e == nullptr) {
        printf("Warning: Null expression in evaluate_expression\n");
        return 0.0;
    }

    // Handle literals directly
    if(e->operand == expr::literal_ee) {
        return e->value;
    }

    // Handle variable references
    if(e->operand == expr::clock_variable_ee) {
        if(e->variable_id < MAX_VARIABLES) {
            return block_state->shared->variables[e->variable_id].value;
        }
        printf("Warning: Invalid variable ID %d in expression\n", e->variable_id);
        return 0.0;
    }

    // Just return the raw value for now
    // TODO: implement full expression evaluation later when basic timing works. We need to support variables i.e. x <= l, where l is not const
    printf("Warning: Non-literal expression (op=%d), using value directly\n",
           e->operand);
    return e->value;
}

__device__ bool check_edge_enabled(const EdgeInfo& edge,
                                 const SharedBlockMemory* shared,
                                 SharedModelState* model,
                                 BlockSimulationState* block_state) {
    printf("\nThread %d: Checking edge %d->%d with %d guards\n",
           threadIdx.x, edge.source_node_id, edge.dest_node_id, edge.num_guards);

    // Check all guards on the edge
    for(int i = 0; i < edge.num_guards; i++) {
        const GuardInfo& guard = model->guards[edge.guards_start_index + i];

        if(guard.uses_variable) {
            int var_id = guard.var_info.variable_id;
            double var_value = shared->variables[var_id].value;
            double bound = evaluate_expression(guard.expression, block_state);

            printf("  Guard %d: var_%d (%s) = %f %s %f\n",
                   i, var_id,
                   guard.var_info.type == VariableKind::CLOCK ? "clock" : "int",
                   var_value,
                   guard.operand == constraint::less_equal_c ? "<=" :
                   guard.operand == constraint::less_c ? "<" :
                   guard.operand == constraint::greater_equal_c ? ">=" :
                   guard.operand == constraint::greater_c ? ">" : "?",
                   bound);

            bool satisfied = false;
            switch(guard.operand) {
                case constraint::less_c:
                    satisfied = var_value < bound; break;
                case constraint::less_equal_c:
                    satisfied = var_value <= bound; break;
                case constraint::greater_c:
                    satisfied = var_value > bound; break;
                case constraint::greater_equal_c:
                    satisfied = var_value >= bound; break;
                default:
                    printf("  Warning: Unknown operator %d\n", guard.operand);
                    return false;
            }

            if(!satisfied) {
                printf("  Guard not satisfied - edge disabled\n");
                return false;
            }
        }
    }

    printf("  All guards satisfied - edge enabled!\n");
    return true;
}

__device__ void check_enabled_edges(ComponentState* my_state,
                                  SharedBlockMemory* shared,
                                  SharedModelState* model,
                                  BlockSimulationState* block_state,
                                  bool is_race_winner) {
    if (!is_race_winner) {
        printf("Thread %d: Skipping edge check (didn't win race)\n", threadIdx.x);
        return;
    }

    printf("\nThread %d: Checking enabled edges for node %d\n",
           threadIdx.x, my_state->current_node->id);

    const NodeInfo& node = *my_state->current_node;
    my_state->num_enabled_edges = 0;  // Reset counter

    // Check each outgoing edge
    for(int i = 0; i < node.num_edges; i++) {
        const EdgeInfo& edge = model->edges[node.first_edge_index + i];
        if(check_edge_enabled(edge, shared, model, block_state)) {
            // Store enabled edge for later selection
            my_state->enabled_edges[my_state->num_enabled_edges++] = i;
            printf("Thread %d: Edge %d is enabled (total enabled: %d)\n",
                   threadIdx.x, i, my_state->num_enabled_edges);
        }
    }

    printf("Thread %d: Found %d enabled edges\n",
           threadIdx.x, my_state->num_enabled_edges);
}




__device__ void check_cuda_error(const char* location) {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error at %s: %s\n", location, hipGetErrorString(error));
    }
}

#define CHECK_ERROR(loc) check_cuda_error(loc)



__device__ void compute_possible_delay(
    ComponentState* my_state,
    SharedBlockMemory* shared,
    SharedModelState* model,
    BlockSimulationState* block_state)
{
    const NodeInfo& node = *my_state->current_node;
    printf("Thread %d: Processing node %d with %d invariants\n",
           threadIdx.x, node.id, node.num_invariants);

    double min_delay = 0.0;
    double max_delay = DBL_MAX;
    bool is_bounded = false;
    if(threadIdx.x == 0) {  // TODO: REMOVE
        for(int i = 0; i < MAX_VARIABLES; i++) {
            if(shared->variables[i].kind == VariableKind::CLOCK) {
                shared->variables[i].rate = 1;
            }
        }
    }

    __syncthreads();

    // Debug current variable values
    printf("Thread %d: Current variable values:\n", threadIdx.x);
    for(int i = 0; i < MAX_VARIABLES; i++) {
        printf("  var[%d] = %f (rate=%d)\n", i,
               shared->variables[i].value,
               shared->variables[i].rate);
    }



    // Process invariants
    for(int i = 0; i < node.num_invariants; i++) {
        const GuardInfo& inv = model->invariants[node.first_invariant_index + i];

        if(inv.uses_variable) {
            int var_id = inv.var_info.variable_id;
            if(var_id >= MAX_VARIABLES) {
                printf("Thread %d: Invalid variable ID %d\n", threadIdx.x, var_id);
                continue;
            }

            auto& var = shared->variables[var_id];
            double current_val = var.value;

            // Set rate to 1 for clocks
            if(inv.var_info.type == VariableKind::CLOCK) {
                var.rate = 1;
            }

            // Evaluate bound expression
            double bound = evaluate_expression(inv.expression, block_state);
            printf("Thread %d: Clock %d invariant: current=%f, bound=%f, rate=%d\n",
                   threadIdx.x, var_id, current_val, bound, var.rate);

            // Only handle upper bounds
            if(inv.operand == constraint::less_c ||
               inv.operand == constraint::less_equal_c) {

                if(var.rate > 0) {  // Only if clock increases
                    double time_to_bound = (bound - current_val) / var.rate;

                    // Add small epsilon for strict inequality
                    if(inv.operand == constraint::less_c) {
                        time_to_bound -= 1e-6;
                    }

                    printf("Thread %d: Computed time_to_bound=%f\n",
                           threadIdx.x, time_to_bound);

                    if(time_to_bound >= 0) {
                        max_delay = min(max_delay, time_to_bound);
                        is_bounded = true;
                        printf("Thread %d: Updated max_delay to %f\n",
                               threadIdx.x, max_delay);
                    }
                }
            }
        }
    }

    // Sample delay if bounded
    if(is_bounded) {
        double rand = hiprand_uniform(block_state->random);
        my_state->next_delay = min_delay + (max_delay - min_delay) * rand;
        my_state->has_delay = true;
        printf("Thread %d: Sampled delay %f in [%f, %f] (rand=%f)\n",
               threadIdx.x, my_state->next_delay, min_delay, max_delay, rand);
    } else {
        printf("Thread %d: No delay bounds, using 1.0\n", threadIdx.x);
        my_state->next_delay = 1.0;  // Default step if no bounds
        my_state->has_delay = true;
    }
}





__device__ double find_minimum_delay(
    ComponentState* my_state,
    SharedBlockMemory* shared,
    SharedModelState* model,
    BlockSimulationState* block_state,
    int num_components)
{
    __shared__ double delays[MAX_COMPONENTS];
    __shared__ int component_indices[MAX_COMPONENTS];

    // Initialize to infinity for inactive threads
    delays[threadIdx.x] = DBL_MAX;
    component_indices[threadIdx.x] = -1;

    // Only active components set their delays
    if(threadIdx.x < num_components && my_state->has_delay) {
        delays[threadIdx.x] = my_state->next_delay;
        component_indices[threadIdx.x] = my_state->component_id;
        printf("Thread %d (component %d): Initial delay %f\n",
               threadIdx.x, my_state->component_id, my_state->next_delay);
    } else {
        printf("Thread %d: Inactive (has_delay=%d, within_components=%d)\n",
               threadIdx.x, my_state->has_delay, threadIdx.x < num_components);
    }
    __syncthreads();

    // Debug print initial state
    if(threadIdx.x == 0) {
        printf("Initial delays: ");
        for(int i = 0; i < num_components; i++) {
            printf("[%d]=%f ", i, delays[i]);
        }
        printf("\n");
    }
    __syncthreads();

    // Find minimum - only active threads participate
    for(int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if(threadIdx.x < stride && threadIdx.x < num_components) {
            int compare_idx = threadIdx.x + stride;
            printf("Thread %d comparing with position %d: %f vs %f\n",
                   threadIdx.x, compare_idx, delays[threadIdx.x],
                   delays[compare_idx]);

            if(delays[compare_idx] < delays[threadIdx.x]) {
                delays[threadIdx.x] = delays[compare_idx];
                component_indices[threadIdx.x] = component_indices[compare_idx];
                printf("Thread %d: Updated minimum to %f from component %d\n",
                       threadIdx.x, delays[threadIdx.x],
                       component_indices[threadIdx.x]);
            }
        }
        __syncthreads();
    }

    // Only thread 0 processes the result
    double min_delay = delays[0];
    if(threadIdx.x == 0) {
        if(min_delay < DBL_MAX) {
            printf("\nFinal result:\n");
            printf("  Minimum delay: %f\n", min_delay);
            printf("  Winning component: %d\n", component_indices[0]);
            printf("\nUpdating clocks:\n");

            // Update all clock values
            for(int i = 0; i < MAX_VARIABLES; i++) {
                if(shared->variables[i].kind == VariableKind::CLOCK) {
                    double old_value = shared->variables[i].value;
                    shared->variables[i].rate = 1;
                    shared->variables[i].value += min_delay;
                    printf("  Clock %d: %f -> %f (advanced by %f)\n",
                           i, old_value, shared->variables[i].value, min_delay);
                }
            }
        } else {
            printf("\nNo valid delays found (all DBL_MAX)\n");
        }
    }
    __syncthreads();

    // Remember who won and check edges only for winner
    bool is_race_winner = (min_delay < DBL_MAX &&
                          component_indices[0] == my_state->component_id);

    if(is_race_winner) {
        printf("\nThread %d (component %d) won race with delay %f\n",
               threadIdx.x, my_state->component_id, min_delay);
    }

    // Check enabled edges only for the winning component
    check_enabled_edges(my_state, shared, model, block_state, is_race_winner);
    __syncthreads();

    return min_delay;
}











int get_total_runs(float confidence, float precision) {
    // confidence level = alpha, i.e. 0.05 for 95% confidence
    // precision = epsilon, i.e. 0.01 for +-1% error

    // int total_runs = (int)ceil(log(2.0/confidence)/log(2.0*precision*precision));
    // int total_runs = static_cast<int>(ceil(log(2.0 / confidence) / log(2.0 * precision * precision)));
    int total_runs = static_cast<size_t>(ceil((log(2.0) - log(confidence)) / (2*pow(precision, 2))));
    return total_runs;
}

// TODO: what if we want to spawn 50 trains? How do we do that?


__global__ void simulation_kernel(SharedModelState* model, bool* results,
                                int runs_per_block, float time_bound) {
    printf("Starting kernel: block=%d, thread=%d\n",
           blockIdx.x, threadIdx.x);
    CHECK_ERROR("kernel start");

    // Verify model pointer
    if(model == nullptr) {
        printf("Thread %d: NULL model pointer!\n", threadIdx.x);
        return;
    }

    __shared__ SharedBlockMemory shared_mem;
    __shared__ ComponentState components[MAX_COMPONENTS];
    __shared__ hiprandState rng_states[MAX_COMPONENTS];

    if (threadIdx.x == 0) {
        // Initialize variables with default values
        for(int i = 0; i < MAX_VARIABLES; i++) {
            shared_mem.variables[i].value = 0.0;
            shared_mem.variables[i].rate = 0;  // Will be set when needed based on guards
            shared_mem.variables[i].kind = VariableKind::INT;  // Default
            shared_mem.variables[i].last_writer = -1;
        }

        // Initialize variables from all invariants in model
        for(int comp = 0; comp < model->num_components; comp++) {
            const NodeInfo& node = model->nodes[comp];
            for(int i = 0; i < node.num_invariants; i++) {
                const GuardInfo& inv = model->invariants[node.first_invariant_index + i];
                if(inv.uses_variable) {
                    int var_id = inv.var_info.variable_id;
                    if(var_id < MAX_VARIABLES) {
                        printf("Block %d: Initializing variable %d from invariant: value=%f, type=%d\n",
                               blockIdx.x, var_id, inv.var_info.initial_value,
                               (int)inv.var_info.type);
                        shared_mem.variables[var_id].value = inv.var_info.initial_value;
                        shared_mem.variables[var_id].kind = inv.var_info.type;
                    }
                }
            }
        }

    }

    __syncthreads();

    CHECK_ERROR("after shared memory declaration");

    // Debug model access
    printf("Thread %d: Attempting to access model, num_components=%d\n",
           threadIdx.x, model->num_components);
    CHECK_ERROR("after model access");

    // Setup block state
    BlockSimulationState block_state;
    block_state.model = model;
    block_state.shared = &shared_mem;
    block_state.my_component = &components[threadIdx.x];

    printf("Thread %d: Block state setup complete\n", threadIdx.x);
    CHECK_ERROR("after block state setup");

    // Initialize RNG
    int sim_id = blockIdx.x * runs_per_block;
    int comp_id = threadIdx.x;
    hiprand_init(1234 + sim_id * blockDim.x + comp_id, 0, 0,
                &rng_states[threadIdx.x]);
    block_state.random = &rng_states[threadIdx.x];

    printf("Thread %d: RNG initialized\n", threadIdx.x);
    CHECK_ERROR("after RNG init");

    // Initialize shared state
    if (threadIdx.x == 0) {
        printf("Block %d: Initializing shared memory\n", blockIdx.x);
        SharedBlockMemory::init(&shared_mem, sim_id);
    }
    __syncthreads();
    CHECK_ERROR("after shared memory init");




    // Initialize component state
    if(threadIdx.x >= model->num_components) {
        printf("Thread %d: Exiting - thread ID exceeds number of components\n",
               threadIdx.x);
        return;
    }

    ComponentState* my_state = block_state.my_component;
    my_state->component_id = comp_id;
    my_state->current_node = &model->nodes[comp_id];
    my_state->has_delay = false;

    printf("Thread %d: Component initialized, node_id=%d\n",
           threadIdx.x, my_state->current_node->id);
    CHECK_ERROR("after component init");

    // Main simulation loop
    while(shared_mem.global_time < time_bound) {
        printf("Thread %d: Time=%f\n", threadIdx.x, shared_mem.global_time);

        compute_possible_delay(my_state, &shared_mem, model, &block_state);
        CHECK_ERROR("after compute delay");
        __syncthreads();

        double min_delay = find_minimum_delay(
    block_state.my_component,  // ComponentState*
    &shared_mem,              // SharedBlockMemory*
    model,                    // SharedModelState*
    &block_state,            // BlockSimulationState*
    model->num_components    // int num_components
);
        CHECK_ERROR("after find minimum");
        printf("Thread %d: Minimum delay = %f\n", threadIdx.x, min_delay);

        if(threadIdx.x == 0) {
            shared_mem.global_time += min_delay;
            printf("Block %d: Advanced time to %f\n",
                   blockIdx.x, shared_mem.global_time);
        }
        __syncthreads();
    }

    printf("Thread %d: Simulation complete\n", threadIdx.x);
}




void simulation::run_statistical_model_checking(SharedModelState* model, float confidence, float precision) {
    int total_runs = 1;
    cout << "total_runs = " << total_runs << endl;

    // Detailed model validation
    if(model == nullptr) {
        cout << "Error: NULL model pointer" << endl;
        return;
    }

    // Print model pointer address
    cout << "Model pointer address: " << model << endl;

    // Try to access model components safely
    hipError_t error;
    SharedModelState host_model;
    error = hipMemcpy(&host_model, model, sizeof(SharedModelState), hipMemcpyDeviceToHost);
    if(error != hipSuccess) {
        cout << "CUDA error copying model: " << hipGetErrorString(error) << endl;
        return;
    }

    cout << "Model components: " << host_model.num_components << endl;

    // Print more model details
    cout << "Component sizes array at: " << host_model.component_sizes << endl;
    cout << "Nodes array at: " << host_model.nodes << endl;
    cout << "Edges array at: " << host_model.edges << endl;

    bool* device_results;
    error = hipMalloc(&device_results, total_runs * sizeof(bool));
    if(error != hipSuccess) {
        cout << "CUDA malloc error: " << hipGetErrorString(error) << endl;
        return;
    }

    // Launch configuration
    int threads_per_block = 2;
    int runs_per_block = 1;
    int num_blocks = 1;

    cout << "Launching kernel with configuration:" << endl;
    cout << "  Blocks: " << num_blocks << endl;
    cout << "  Threads per block: " << threads_per_block << endl;
    cout << "  Time bound: " << TIME_BOUND << endl;

    // Launch kernel
    simulation_kernel<<<num_blocks, threads_per_block>>>(
        model, device_results, runs_per_block, TIME_BOUND);

    error = hipGetLastError();
    if(error != hipSuccess) {
        cout << "Kernel launch error: " << hipGetErrorString(error) << endl;
        return;
    }

    cout << "Kernel launched successfully, waiting for completion..." << endl;

    error = hipDeviceSynchronize();
    if(error != hipSuccess) {
        cout << "Kernel execution error: " << hipGetErrorString(error) << endl;
        return;
    }

    cout << "Kernel execution complete" << endl;
}






__global__ void findSmallestElementInArray(float *input, int input_length, float *result, int nblocks) {
    int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    int nThreads = blockDim.x * nblocks;
    for (int i = 0; i < ceil(log2f(nThreads)); i++) { //1
        if (threadid % static_cast<int>(pow(2, i+1)==0)) {
            int correspondant = min(static_cast<int>(threadid + pow(2, i)), input_length-1);
            input[threadid] = min(input[threadid], input[correspondant]);
        }
    }
    if (threadid == 0) {
        *result = input[0];
    }
}

// void testFunction () {
//     float* h_a = new float[NUM_RUNS];
//     srand( static_cast<unsigned>(time(NULL)));
//     int upper = 3500;
//     int lower = 1230;
//     for (int i = 0; i < NUM_RUNS; i++) {
//
//         h_a[i] = rand() % (upper - lower) + lower;
//         cout << h_a[i] << ", ";
//     }
//     cout << endl;
//
//     float* d_a;
//     float d_result;
//     hipMalloc(&d_a, NUM_RUNS * sizeof(float));
//
//     hipMemcpy(d_a, h_a, NUM_RUNS * sizeof(float), hipMemcpyHostToDevice);
//
//     findSmallestElementInArray<<<1, 128>>>(d_a, NUM_RUNS, &d_result, 1); // 2 blocks (component size), 100 simulations but round up to 128
//
//     hipMemcpy(h_a, d_a, NUM_RUNS * sizeof(float), hipMemcpyDeviceToHost);
//
//     for (int i = 0; i < NUM_RUNS; i++) {
//         cout << h_a[i] << ", ";
//     }
//     cout <<endl << "Result = " << d_result << endl;
// }

void simulation::runSimulation() {
    // Problem with models, spawning new components Trains in train gate for example?
    // componentSimulation<<<NUM_COMPONENTS, 128>>>(); // 2 blocks (component size), 100 simulations but round up to 128

    // testFunction();

    // Pick delays: implement delay function
    // Find the smallest delay, and which index it has (to find component it belongs to)
    // Apply the delay
    // Pick a transition from the component that won: Pick according to the weights
    // Check whether we need to synchronize with anything when taking this transition
    // Take the transition

    // We need the state such that we can describe the run afterwards. We add our delays to it.



    cout << "test from run sim" << endl;
}
