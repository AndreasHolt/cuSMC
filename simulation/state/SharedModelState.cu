#include "hip/hip_runtime.h"
//
// Created by andwh on 04/11/2024.
//

#include "SharedModelState.cuh"

#include <iostream>

#define MAX_NODES_PER_COMPONENT 5

void count_edges_and_constraints(
    const std::vector<std::vector<node*>>& components_nodes,
    int& total_edges,
    int& total_guards,
    int& total_updates,
    std::vector<int>& edges_per_node,
    std::vector<int>& node_edge_starts)
{
    total_edges = 0;
    total_guards = 0;
    total_updates = 0;

    // For each node level
    for(int node_idx = 0; node_idx < MAX_NODES_PER_COMPONENT; node_idx++) {
        // For each component
        for(int comp_idx = 0; comp_idx < components_nodes.size(); comp_idx++) {
            if(node_idx < components_nodes[comp_idx].size()) {
                node* current_node = components_nodes[comp_idx][node_idx];

                // Store start index for this node's edges
                node_edge_starts.push_back(total_edges);
                edges_per_node.push_back(current_node->edges.size);

                // Count guards and updates for each edge
                for(int e = 0; e < current_node->edges.size; e++) {
                    total_edges++;
                    total_guards += current_node->edges[e].guards.size;
                    total_updates += current_node->edges[e].updates.size;
                }
            }
        }
    }
}

void fill_edge_arrays(
    const std::vector<std::vector<node*>>& components_nodes,
    std::vector<EdgeInfo>& host_edges,
    std::vector<GuardInfo>& host_guards,
    std::vector<UpdateInfo>& host_updates)
{
    int current_guard_index = 0;
    int current_update_index = 0;

    // For each node level
    for(int node_idx = 0; node_idx < MAX_NODES_PER_COMPONENT; node_idx++) {
        // For each component
        for(int comp_idx = 0; comp_idx < components_nodes.size(); comp_idx++) {
            if(node_idx < components_nodes[comp_idx].size()) {
                node* current_node = components_nodes[comp_idx][node_idx];

                // Add edges for this node in coalesced layout
                for(int e = 0; e < current_node->edges.size; e++) {
                    const edge& current_edge = current_node->edges[e];

                    // Store edge info
                    host_edges.emplace_back(
                        current_node->id,              // source_node_id
                        current_edge.dest->id,         // dest_node_id
                        current_edge.channel,          // channel
                        current_edge.weight,           // weight
                        current_edge.guards.size,      // num_guards
                        current_guard_index,           // guards_start_index
                        current_edge.updates.size,     // num_updates
                        current_update_index           // updates_start_index
                    );

                    // Store guards
                    for(int g = 0; g < current_edge.guards.size; g++) {
                        const constraint& guard = current_edge.guards[g];
                        host_guards.push_back(GuardInfo{
                            guard.operand,
                            guard.uses_variable,
                            guard.value,
                            guard.expression
                        });
                        current_guard_index++;
                    }

                    // Store updates
                    for(int u = 0; u < current_edge.updates.size; u++) {
                        const update& upd = current_edge.updates[u];
                        host_updates.push_back(UpdateInfo{
                            upd.variable_id,
                            upd.expression
                        });
                        current_update_index++;
                    }
                }
            }
        }
    }
}





void print_node_info(const node* n, const std::string& prefix = "") {
    std::cout << prefix << "Node ID: " << n->id << " Type: " << n->type << "\n";
    std::cout << prefix << "Edges:\n";
    for(int i = 0; i < n->edges.size; i++) {
        const edge& e = n->edges[i];
        std::cout << prefix << "  -> Dest ID: " << e.dest->id
                 << " Channel: " << e.channel << "\n";
    }
}



SharedModelState* init_shared_model_state(
   const network* cpu_network,
   const std::unordered_map<int, int>& node_subsystems_map,
   const std::unordered_map<int, std::list<edge>>& node_edge_map, const std::unordered_map<int, node*>& node_map)
{
   // First organize nodes by component
   std::vector<std::vector<std::pair<int, const std::list<edge>*>>> components_nodes;
   int max_component_id = -1;

   // Find number of components
   for(const auto& pair : node_subsystems_map) {
       max_component_id = std::max(max_component_id, pair.second);
   }
   components_nodes.resize(max_component_id + 1);

   // Group nodes by component
   for(const auto& pair : node_edge_map) {
       int node_id = pair.first;
       const std::list<edge>& edges = pair.second;
       int component_id = node_subsystems_map.at(node_id);
       components_nodes[component_id].push_back({node_id, &edges});
   }

   // Find max nodes per component for array sizing
   int max_nodes_per_component = 0;
   std::vector<int> component_sizes(components_nodes.size());
   for(int i = 0; i < components_nodes.size(); i++) {
       component_sizes[i] = components_nodes[i].size();
       max_nodes_per_component = std::max(max_nodes_per_component,
                                        component_sizes[i]);
   }

   // Allocate device memory for component sizes
   int* device_component_sizes;
   hipMalloc(&device_component_sizes,
              components_nodes.size() * sizeof(int));
   hipMemcpy(device_component_sizes, component_sizes.data(),
              components_nodes.size() * sizeof(int),
              hipMemcpyHostToDevice);

   // Count total edges, guards, updates and invariants
   int total_edges = 0;
   int total_guards = 0;
   int total_updates = 0;
   int total_invariants = 0;
   for(const auto& pair : node_edge_map) {
       // Count edges, guards and updates
       for(const auto& edge : pair.second) {
           total_edges++;
           total_guards += edge.guards.size;
           total_updates += edge.updates.size;
       }

       // Count invariants from nodes
       node* current_node = node_map.at(pair.first);  // Now we can get the node!
       total_invariants += current_node->invariants.size;
   }

   // Allocate device memory
   const int total_node_slots = max_nodes_per_component * components_nodes.size();
   NodeInfo* device_nodes;
   EdgeInfo* device_edges;
   GuardInfo* device_guards;
   UpdateInfo* device_updates;
   GuardInfo* device_invariants;
   hipMalloc(&device_nodes, total_node_slots * sizeof(NodeInfo));
   hipMalloc(&device_edges, total_edges * sizeof(EdgeInfo));
   hipMalloc(&device_guards, total_guards * sizeof(GuardInfo));
   hipMalloc(&device_updates, total_updates * sizeof(UpdateInfo));
   hipMalloc(&device_invariants, total_invariants * sizeof(GuardInfo));

   // Create host arrays
   std::vector<NodeInfo> host_nodes;
   std::vector<EdgeInfo> host_edges;
   std::vector<GuardInfo> host_guards;
   std::vector<UpdateInfo> host_updates;
   std::vector<GuardInfo> host_invariants;
   host_nodes.reserve(total_node_slots);
   host_edges.reserve(total_edges);
   host_guards.reserve(total_guards);
   host_updates.reserve(total_updates);
   host_invariants.reserve(total_invariants);

   int current_edge_index = 0;
   int current_guard_index = 0;
   int current_update_index = 0;
   int current_invariant_index = 0;

   // For each node level
   for(int node_idx = 0; node_idx < max_nodes_per_component; node_idx++) {
       // For each component at this level
       for(int comp_idx = 0; comp_idx < components_nodes.size(); comp_idx++) {
           if(node_idx < components_nodes[comp_idx].size()) {
               const auto& node_pair = components_nodes[comp_idx][node_idx];
               int node_id = node_pair.first;
               const std::list<edge>& edges = *node_pair.second;
               node* current_node = node_map.at(node_id);

               // Store invariants
               int invariants_start = current_invariant_index;
               for(int i = 0; i < current_node->invariants.size; i++) {
                   const constraint& inv = current_node->invariants[i];
                   host_invariants.push_back(GuardInfo{
                       inv.operand,
                       inv.uses_variable,
                       inv.value,
                       inv.expression
                   });
                   current_invariant_index++;
               }

               // Create NodeInfo with edge and invariant information
               NodeInfo node_info{
                   node_id,                    // id
                   current_node->type,         // type (now using actual node type)
                   node_idx,                   // level
                   current_node->lamda,        // lambda (using node's lambda)
                   current_edge_index,         // first_edge_index
                   static_cast<int>(edges.size()), // num_edges
                   invariants_start,           // first_invariant_index
                   static_cast<int>(current_node->invariants.size) // num_invariants
               };
               host_nodes.push_back(node_info);

               // Add edges and their guards/updates
               for(const edge& e : edges) {
                   // Store guards
                   int guards_start = current_guard_index;
                   for(int g = 0; g < e.guards.size; g++) {
                       const constraint& guard = e.guards[g];
                       host_guards.push_back(GuardInfo{
                           guard.operand,
                           guard.uses_variable,
                           guard.value,
                           guard.expression
                       });
                       current_guard_index++;
                   }

                   // Store updates
                   int updates_start = current_update_index;
                   for(int u = 0; u < e.updates.size; u++) {
                       const update& upd = e.updates[u];
                       host_updates.push_back(UpdateInfo{
                           upd.variable_id,
                           upd.expression
                       });
                       current_update_index++;
                   }

                   // Create edge info
                   EdgeInfo edge_info{
                       node_id,
                       e.dest->id,
                       e.channel,
                       e.weight,
                       e.guards.size,
                       guards_start,
                       e.updates.size,
                       updates_start
                   };
                   host_edges.push_back(edge_info);
                   current_edge_index++;
               }
           } else {
               // Padding for components with fewer nodes
               host_nodes.push_back(NodeInfo{
                   -1,                 // id
                   node::location,     // type
                   -1,                 // level
                   nullptr,            // lambda
                   -1,                 // first_edge_index
                   0,                  // num_edges
                   -1,                 // first_invariant_index
                   0                   // num_invariants
               });
           }
       }
   }

   // Copy everything to device
   hipMemcpy(device_nodes, host_nodes.data(),
              total_node_slots * sizeof(NodeInfo),
              hipMemcpyHostToDevice);
   hipMemcpy(device_edges, host_edges.data(),
              total_edges * sizeof(EdgeInfo),
              hipMemcpyHostToDevice);
   hipMemcpy(device_guards, host_guards.data(),
              total_guards * sizeof(GuardInfo),
              hipMemcpyHostToDevice);
   hipMemcpy(device_updates, host_updates.data(),
              total_updates * sizeof(UpdateInfo),
              hipMemcpyHostToDevice);
   hipMemcpy(device_invariants, host_invariants.data(),
              total_invariants * sizeof(GuardInfo),
              hipMemcpyHostToDevice);

   // Create and copy SharedModelState
   SharedModelState host_model{
       static_cast<int>(components_nodes.size()),
       device_component_sizes,
       device_nodes,
       device_edges,
       device_guards,
       device_updates,
       device_invariants    // Add invariants to constructor
   };

   SharedModelState* device_model;
   hipMalloc(&device_model, sizeof(SharedModelState));
   hipMemcpy(device_model, &host_model, sizeof(SharedModelState),
              hipMemcpyHostToDevice);

   return device_model;
}







__global__ void test_kernel(SharedModelState* model) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Number of components: %d\n", model->num_components);

        // Print nodes in coalesced layout
        for(int node_idx = 0; node_idx < model->component_sizes[0]; node_idx++) {
            printf("\nNode level %d:\n", node_idx);

            for(int comp = 0; comp < model->num_components; comp++) {
                if(node_idx < model->component_sizes[comp]) {
                    const NodeInfo& node = model->nodes[node_idx * model->num_components + comp];

                    // Skip padding nodes
                    if(node.id == -1) continue;

                    // Print node info
                    if(node.num_edges > 0) {
                        printf("Component %d: ID=%d, Type=%d (Edges: %d-%d)\n",
                               comp, node.id, node.type,
                               node.first_edge_index,
                               node.first_edge_index + node.num_edges - 1);
                    } else {
                        printf("Component %d: ID=%d, Type=%d (No edges)\n",
                               comp, node.id, node.type);
                    }


                    // Now use direct edge indexing
                    for(int e = 0; e < node.num_edges; e++) {
                        const EdgeInfo& edge = model->edges[node.first_edge_index + e];
                        printf("  Edge %d: %d -> %d (channel: %d)\n",
                               node.first_edge_index + e, edge.source_node_id,
                               edge.dest_node_id, edge.channel);

                        // Print guards
                        printf("    Guards (%d):\n", edge.num_guards);
                        for(int g = 0; g < edge.num_guards; g++) {
                            const GuardInfo& guard = model->guards[edge.guards_start_index + g];
                            printf("      Guard %d: op=%d, uses_var=%d\n",
                                   g, guard.operand, guard.uses_variable);
                        }

                        // Print updates
                        printf("    Updates (%d):\n", edge.num_updates);
                        for(int u = 0; u < edge.num_updates; u++) {
                            const UpdateInfo& update = model->updates[edge.updates_start_index + u];
                            printf("      Update %d: var_id=%d\n",
                                   u, update.variable_id);
                        }
                    }
                }
            }
        }
    }
}



__global__ void validate_edge_indices(SharedModelState* model) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int total_edges = 0;
        bool indices_valid = true;

        // Validate each node's edge indices
        for(int level = 0; level < model->component_sizes[0]; level++) {
            for(int comp = 0; comp < model->num_components; comp++) {
                if(level < model->component_sizes[comp]) {
                    const NodeInfo& node = model->nodes[level * model->num_components + comp];
                    if(node.id != -1) {
                        // Check edge indices are in range
                        if(node.first_edge_index < 0) {
                            printf("Error: Node %d has negative edge index\n", node.id);
                            indices_valid = false;
                        }

                        // Check edges are sequential
                        if(node.first_edge_index < total_edges) {
                            printf("Error: Node %d edges overlap with previous node\n", node.id);
                            indices_valid = false;
                        }

                        // Verify all edges belong to this node
                        for(int e = 0; e < node.num_edges; e++) {
                            const EdgeInfo& edge = model->edges[node.first_edge_index + e];
                            if(edge.source_node_id != node.id) {
                                printf("Error: Edge %d doesn't belong to node %d\n",
                                       node.first_edge_index + e, node.id);
                                indices_valid = false;
                            }
                        }

                        total_edges += node.num_edges;
                    }
                }
            }
        }

        printf("\n");
        printf("==========================================\n");
        printf("Edge index validation %s\n", indices_valid ? "PASSED" : "FAILED");
        printf("Total edges: %d\n", total_edges);
        printf("==========================================\n");
    }
}


__global__ void verify_invariants_kernel(SharedModelState* model) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {  // Single thread for debugging output
        printf("\nVerifying Invariants:\n");
        printf("==========================================\n");

        // For each node level
        for(int node_idx = 0; node_idx < model->component_sizes[0]; node_idx++) {
            printf("\nNode Level %d:\n", node_idx);

            // For each component
            for(int comp = 0; comp < model->num_components; comp++) {
                if(node_idx < model->component_sizes[comp]) {
                    const NodeInfo& node = model->nodes[node_idx * model->num_components + comp];

                    // Skip padding nodes
                    if(node.id == -1) continue;

                    printf("\nComponent %d, Node ID %d:\n", comp, node.id);
                    printf("  Type: %d\n", node.type);
                    printf("  Invariants: %d\n", node.num_invariants);

                    // Print each invariant
                    for(int i = 0; i < node.num_invariants; i++) {
                        const GuardInfo& inv = model->invariants[node.first_invariant_index + i];
                        printf("    Invariant %d:\n", i);
                        printf("      Operator: %d\n", inv.operand);
                        printf("      Uses Variable: %d\n", inv.uses_variable);
                        // Can add more detailed invariant info here
                    }

                    // Verify indices are valid
                    if(node.num_invariants > 0) {
                        if(node.first_invariant_index < 0) {
                            printf("ERROR: Invalid invariant index %d for node %d\n",
                                   node.first_invariant_index, node.id);
                        }
                    }
                }
            }
        }
        printf("\n==========================================\n");
    }
}





