#include "hip/hip_runtime.h"
//
// Created by andreas on 12/11/24.
//

#include "smc.cuh"
#include "main.cuh"
#include "simulation/write_csv.cuh"
#define TIME_BOUND 10.0


__host__ void run_statistical_model_checking(SharedModelState *model, float confidence, float precision,
                                                VariableKind *kinds, bool* flags,
                                                double* variable_flags, model_info m_info, configuration conf, statistics_Configuration stat_conf) {
    // Get max components from SharedModelState
    int MC;
    hipMemcpy(&MC, &(model->num_components), sizeof(int), hipMemcpyDeviceToHost);

    if constexpr (VERBOSE) {
        cout << "total_runs = " << stat_conf.simulations << endl;
    }
    // Validate parameters
    if (model == nullptr) {
        cout << "Error: NULL model pointer" << endl;
        return;
    }

    // Get device properties and validate configuration
    hipDeviceProp_t deviceProp;
    hipError_t error = hipGetDeviceProperties(&deviceProp, 0);
    if (error != hipSuccess) {
        cout << "Error getting device properties: " << hipGetErrorString(error) << endl;
        return;
    }

    // Adjust threads to be multiple of warp size
    int warp_size = deviceProp.warpSize;
    //int threads_per_block = 512; // 100 components
    // int threads_per_block = ((2 + warp_size - 1) / warp_size) * warp_size; // Round up to nearest warp
    int threads_per_block = 32; // 100 components
    int runs_per_block = m_info.runs_per_block;
    int num_blocks = stat_conf.simulations;

    // Print detailed device information
    if constexpr (VERBOSE) {
        cout << "Device details:" << endl
                << "  Name: " << deviceProp.name << endl
                << "  Warp size: " << warp_size << endl
                << "  Max threads per block: " << deviceProp.maxThreadsPerBlock << endl
                << "  Max block dimensions: " << deviceProp.maxThreadsDim[0] << " x "
                << deviceProp.maxThreadsDim[1] << " x " << deviceProp.maxThreadsDim[2] << endl
                << "  Adjusted threads per block: " << threads_per_block << endl;
    }

    // Validate configuration
    if (threads_per_block > deviceProp.maxThreadsPerBlock) {
        cout << "Error: threads_per_block (" << threads_per_block
                << ") exceeds device maximum (" << deviceProp.maxThreadsPerBlock << ")" << endl;
        return;
    }

    if (num_blocks > deviceProp.maxGridSize[0]) {
        cout << "Error: num_blocks (" << num_blocks
                << ") exceeds device maximum (" << deviceProp.maxGridSize[0] << ")" << endl;
        return;
    }

    // Verify shared memory size is sufficient
    size_t shared_mem_per_block = sizeof(SharedBlockMemory);
    if (shared_mem_per_block > deviceProp.sharedMemPerBlock) {
        cout << "Error: Required shared memory (" << shared_mem_per_block
                << ") exceeds device capability (" << deviceProp.sharedMemPerBlock << ")" << endl;
        return;
    }

    // Allocate and validate device results array
    bool *device_results;
    error = hipMalloc(&device_results, stat_conf.simulations * sizeof(bool));
    if (error != hipSuccess) {
        cout << "CUDA malloc error: " << hipGetErrorString(error) << endl;
        return;
    }

    if constexpr (VERBOSE) {
        cout << "Launch configuration validated:" << endl;
        cout << "  Blocks: " << num_blocks << endl;
        cout << "  Threads per block: " << threads_per_block << endl;
        cout << "  Shared memory per block: " << shared_mem_per_block << endl;
        cout << "  Time bound: " << stat_conf.timeBound << endl;
    }

    // Verify model is accessible
    SharedModelState host_model;
    error = hipMemcpy(&host_model, model, sizeof(SharedModelState), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        cout << "Error copying model: " << hipGetErrorString(error) << endl;
        hipFree(device_results);
        return;
    }
    if constexpr (VERBOSE) {
        cout << "Model verified accessible with " << host_model.num_components << " components" << endl;
    }
    // Add verification here with more safety checks
    if constexpr (VERBOSE) {
        cout << "\nVerifying model transfer:" << endl;
        cout << "Model contents:" << endl;
        cout << "  nodes pointer: " << host_model.nodes << endl;
        cout << "  invariants pointer: " << host_model.invariants << endl;
        cout << "  num_components: " << host_model.num_components << endl;
    }
    if (host_model.nodes == nullptr) {
        cout << "Error: Nodes array is null" << endl;
        hipFree(device_results);
        return;
    }

    // Try to read just the pointer first
    void *nodes_ptr;
    error = hipMemcpy(&nodes_ptr, (void *) &(model->nodes), sizeof(void *), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        cout << "Error reading nodes pointer: " << hipGetErrorString(error) << endl;
        hipFree(device_results);
        return;
    }
    if constexpr (VERBOSE) {
        cout << "Nodes pointer verification: " << nodes_ptr << endl;
    }
    // Now try to read one node
    NodeInfo test_node;
    error = hipMemcpy(&test_node, host_model.nodes, sizeof(NodeInfo), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        cout << "Error reading node: " << hipGetErrorString(error) << endl;
        hipFree(device_results);
        return;
    }
    if constexpr (VERBOSE) {
        cout << "First node verification:" << endl
                << "  ID: " << test_node.id << endl
                << "  First invariant index: " << test_node.first_invariant_index << endl
                << "  Num invariants: " << test_node.num_invariants << endl;
    }
    // Only check invariants if we have a valid pointer
    if (host_model.invariants != nullptr) {
        if constexpr (VERBOSE) {
            cout << "Attempting to read invariant..." << endl;
        }
        GuardInfo test_guard;
        error = hipMemcpy(&test_guard, host_model.invariants, sizeof(GuardInfo),
                           hipMemcpyDeviceToHost);
        if (error != hipSuccess) {
            cout << "Error reading invariant: " << hipGetErrorString(error) << endl;
            hipFree(device_results);
            return;
        }
        if constexpr (VERBOSE) {
            cout << "First invariant verification:" << endl
                    << "  Uses variable: " << test_guard.uses_variable << endl
                    << "  Variable ID: " << (test_guard.uses_variable ? test_guard.var_info.variable_id : -1) << endl;
        }
    } else {
        cout << "No invariants pointer available" << endl;
    }


    // Check each kernel parameter
    if constexpr (VERBOSE) {
        cout << "Kernel parameter validation:" << endl;
        cout << "  model pointer: " << model << endl;
        cout << "  device_results pointer: " << device_results << endl;
        cout << "  runs_per_block: " << runs_per_block << endl;
        cout << "  TIME_BOUND: " << TIME_BOUND << endl;
    }

    // Verify model pointer is a valid device pointer
    hipPointerAttribute_t modelAttrs;
    error = hipPointerGetAttributes(&modelAttrs, model);
    if (error != hipSuccess) {
        cout << "Error checking model pointer: " << hipGetErrorString(error) << endl;
        hipFree(device_results);
        return;
    }
    if constexpr (VERBOSE) {
        cout << "Model pointer properties:" << endl;
        cout << "  type: " << (modelAttrs.type == hipMemoryTypeDevice ? "device" : "other") << endl;
        cout << "  device: " << modelAttrs.device << endl;
    }

    // Similarly check device_results pointer
    hipPointerAttribute_t resultsAttrs;
    error = hipPointerGetAttributes(&resultsAttrs, device_results);
    if (error != hipSuccess) {
        cout << "Error checking results pointer: " << hipGetErrorString(error) << endl;
        hipFree(device_results);
        return;
    }
    if constexpr (VERBOSE) {
        cout << "Results pointer properties:" << endl;
        cout << "  type: " << (resultsAttrs.type == hipMemoryTypeDevice ? "device" : "other") << endl;
        cout << "  device: " << resultsAttrs.device << endl;
    }
    // Clear any previous error
    error = hipGetLastError();
    if (error != hipSuccess) {
        cout << "Previous error cleared: " << hipGetErrorString(error) << endl;
    }

    VariableKind *d_kinds;
    error = hipMalloc(&d_kinds, m_info.num_vars * sizeof(VariableKind)); // Assuming MAX_VARIABLES is defined
    if (error != hipSuccess) {
        cout << "CUDA malloc error for kinds array: " << hipGetErrorString(error) << endl;
        return;
    }

    error = hipMemcpy(d_kinds, kinds, m_info.num_vars * sizeof(VariableKind), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        cout << "Error copying kinds array: " << hipGetErrorString(error) << endl;
        hipFree(d_kinds);
        return;
    }
    if constexpr (VERBOSE) {
        cout << "Launching kernel..." << endl;
    }


    // RNG States
    hiprandState *rng_states_global;

    if constexpr (USE_GLOBAL_MEMORY_CURAND) {
        hipMalloc(&rng_states_global, MC * sizeof(hiprandState));
    }


    hipGetDeviceProperties(&deviceProp, 0); // Assuming device 0, change if necessary



    // What share memory we need:
    /*
    __shared__ double delays[MAX_COMPONENTS]; // Only need MAX_COMPONENTS slots, not full warp size
    __shared__ int component_indices[MAX_COMPONENTS];
    __shared__ SharedBlockMemory shared_mem;
    __shared__ ComponentState components[MAX_COMPONENTS];
    hiprandState *rng_states;

    // Store curandStates in either global memory or shared memory. Requires ~90kb of shared memory w/ curandStates
    if constexpr (USE_GLOBAL_MEMORY_CURAND) {
        // extern __shared__ hiprandState *rng_states_global;
        rng_states = rng_states_global;
    } else {
        __shared__ hiprandState rng_states_shared[MAX_COMPONENTS];
        rng_states = rng_states_shared;
    }



    */

    const chrono::steady_clock::time_point global_start = chrono::steady_clock::now();
    // Dynamic Shared memory: https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/
    // int MC = m_info.MAX_COMPONENTS;
    if constexpr (USE_GLOBAL_MEMORY_CURAND) {
        simulation_kernel<<<num_blocks, threads_per_block, MC*sizeof(double) + MC*sizeof(int) + sizeof(SharedBlockMemory) + MC*sizeof(ComponentState)>>>(
        model, device_results, runs_per_block, stat_conf.timeBound, d_kinds, m_info.num_vars, flags, variable_flags, stat_conf.variable_id, stat_conf.isMax, rng_states_global, conf.curand_seed, MC);
    } else {
        simulation_kernel<<<num_blocks, threads_per_block, MC*sizeof(double) + MC*sizeof(int) + sizeof(SharedBlockMemory) + MC*sizeof(ComponentState) + MC*sizeof(hiprandState)>>>(
        model, device_results, runs_per_block, stat_conf.timeBound, d_kinds, m_info.num_vars, flags, variable_flags, stat_conf.variable_id, stat_conf.isMax, rng_states_global, conf.curand_seed, MC);
    }

    hipDeviceSynchronize();




    auto duration_nano = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::steady_clock::now() - global_start);
    long long nanoseconds = duration_nano.count();
    double ms = duration_nano.count() / 1000000.0;
    cout << "Time taken: " << ms << " ms on file " << conf.filename << endl;
    cout << "Time taken: " << nanoseconds << " on file " << conf.filename << endl;
    std::cout << "Time taken: " << duration_nano.count() << " ns ("
          << (duration_nano.count() / 1000000.0) << " ms)" << std::endl;

    writeTimingToCSV(conf.filename, MC, stat_conf.simulations, stat_conf.timeBound, ms);



    // Check for launch error
    error = hipGetLastError();
    if (error != hipSuccess) {
        cout << "Launch error: " << hipGetErrorString(error) << endl;
        hipFree(device_results);
        return;
    }

    // Check for execution error
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        cout << "Execution error: " << hipGetErrorString(error) << endl;
        hipFree(device_results);
        return;
    }
    if constexpr (VERBOSE) {
        cout << "Kernel completed successfully" << endl;
    }

    // Cleanup
    hipFree(d_kinds);
    hipFree(device_results);
}

void smc(configuration conf, statistics_Configuration stat_conf) {
    // Read and parse XML file
    abstract_parser *parser = new uppaal_xml_parser();
    network model = parser->parse(conf.filename);
    network_props properties = {};
    populate_properties(properties, parser);

    std::unordered_set<std::string> *query_set = new std::unordered_set<std::string>();
    query_set->insert(stat_conf.loc_query);
    // Optimize the model
    domain_optimization_visitor optimizer = domain_optimization_visitor(
        query_set,
        properties.node_network,
        properties.node_names,
        properties.template_names);
    optimizer.optimize(&model);

    // Compile expressions to PN
    pn_compile_visitor pn_compiler;
    pn_compiler.visit(&model);

    // Gather relevant information about variables
    VariableTrackingVisitor var_tracker;
    var_tracker.visit(&model);

    auto registry = var_tracker.get_variable_registry();

    if (PRINT_VARIABLES) {
        auto node_name = parser->get_nodes_with_name();
        auto variable_name = parser->get_variables_names_to_ids_map();
        for (auto iter = node_name->begin(); iter != node_name->end(); ++iter) {
            auto relation = *iter;
            cout << "Node id: " << relation.first << " name: " << relation.second << endl;
        }
        for (auto iter = variable_name.begin(); iter != variable_name.end(); ++iter) {
            auto relation = *iter;
            cout << "Variable id: " << relation.first << " name: " << relation.second << endl;
        }


    }

    VariableKind *kinds = var_tracker.createKindArray(registry);
    uint32_t num_vars = registry.size();

    // TODO: Calculate these values... (MAX VALUE STACK = FANOUT?
    const struct model_info m_info = { 64, 1, num_vars};


    cout << "=================\n\n";
    cout << "Running SMC with the following settings:" << std::endl;
    cout << "- Number of simulations: " << stat_conf.simulations << std::endl;
    cout << "- Model: " << conf.filename << std::endl;
    cout << "- Random seed: " << conf.curand_seed << std::endl;
    cout << "- Time bound: " << stat_conf.timeBound << std::endl;

    if (!stat_conf.loc_query.empty()) {
        cout << "- Logging type: \"comp.node\" query" << std::endl;
    } else if (stat_conf.variable_id != -1) {
        string min_or_max = (stat_conf.isMax) ? "max" : "min";
        cout << "- Logging type: variable query. Finding " << min_or_max
             << " of variable with ID " << stat_conf.variable_id << std::endl;
    }

    cout << "=================\n\n";

    double result = 0;
    // Handling variable queries
    if (stat_conf.variable_id != -1) {
        std::unordered_map<int, node *> node_map = optimizer.get_node_map();
        SharedModelState *state = init_shared_model_state(
            &model, // cpu_network
            *optimizer.get_node_subsystems_map(),
            *properties.node_edge_map,
            node_map,
            var_tracker.get_variable_registry(),
            parser,
            m_info.num_vars);
        Statistics stats(stat_conf.simulations, VAR_STAT);


        run_statistical_model_checking(state, 0.05, 0.01, kinds, stats.get_comp_device_ptr(),
                                           stats.get_var_device_ptr(), m_info, conf, stat_conf);

        int len_of_array = stat_conf.simulations;
        double *var_data = stats.collect_var_data();
        // Estimate query
        if (stat_conf.isEstimate) {
            for (int i = 0; i < len_of_array; i++) {
                cout << "Adding " << var_data[i] << " to sum " << result << endl;
                result += var_data[i];
            }
            result = result / len_of_array;
        }
        // Probability query
        else if (!stat_conf.isEstimate) {
            for (int i = 0; i < len_of_array; i++) {
                if (stat_conf.isMax && var_data[i] > stat_conf.variable_threshhold) { // Increment if value is larger than specified max
                    result += 1;
                }
                if (!stat_conf.isMax && var_data[i] < stat_conf.variable_threshhold) { // Increment if value is smaller than specified min
                    result += 1;
                }
            }
            result = result / len_of_array;
        }
        std::cout << "Result: " << result << endl;
    }

    if (stat_conf.variable_id == -1) {
        Statistics stats(stat_conf.simulations, COMP_STAT);
        if constexpr (VERBOSE) {
            cout << "Recorded query is: " + stat_conf.loc_query << endl;
        }

        // String split
        std::vector<char> component;
        std::vector<char> goal_node;
        bool period_reached = false;
        for (int i = 0; i < stat_conf.loc_query.length(); i++) {
            // Guard
            if (stat_conf.loc_query[i] == '.') {
                period_reached = true;
                continue;
            }
            if (!period_reached) {
                component.push_back(stat_conf.loc_query[i]);
            }
            if (period_reached) {
                goal_node.push_back(stat_conf.loc_query[i]);
            }
        }

        std::string component_name(component.begin(), component.end());
        std::string node_name(goal_node.begin(), goal_node.end());
        auto temp = properties.node_name_int_map.find(node_name);
        std::unordered_map<int, node *> node_map = optimizer.get_node_map();

        if (temp != properties.node_name_int_map.cend()) {
            int goal_node_idx = (*temp).second;

            (*node_map.find(goal_node_idx)).second->type = node::goal;
        }

        SharedModelState *state = init_shared_model_state(
            &model, // cpu_network
            *optimizer.get_node_subsystems_map(),
            *properties.node_edge_map,
            node_map,
            var_tracker.get_variable_registry(),
            parser,
            num_vars);

        // Run the SMC simulations
        run_statistical_model_checking(state, 0.05, 0.01, kinds, stats.get_comp_device_ptr(),
                                           stats.get_var_device_ptr(), m_info, conf, stat_conf);
        try {
            auto results = stats.collect_results();
            stats.print_results(stat_conf.loc_query, results);
        } catch (const std::runtime_error &e) {
            cout << "Error while collecting the results from the simulations: " << e.what() << endl;
            }

    }
    // Kernels for debugging purposes
    if constexpr (VERBOSE) {
        // verify_expressions_kernel<<<1,1>>>(state);
        // test_kernel<<<1, 1>>>(state);
        // validate_edge_indices<<<1, 1>>>(state);
    }

    delete parser;
}
